#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <chrono>

#include "KDTree.hpp"

const int numberOfPoints = 4;
const int numbersPerThread = 2;

typedef std::chrono::high_resolution_clock Time;
typedef std::chrono::milliseconds ms;
typedef std::chrono::duration<float> fsec;

__device__ void swap(float &a, float &b)
{
	float c = a;
	a = b;
	b = c;
}

__device__ void swap(int &a, int &b)
{
	int c = a;
	a = b;
	b = c;
}

__device__ void printArray(int * a, int size)
{
	for (int i = 0; i < size; i++) {
		printf("%d ", a[i]);
	}
	printf("\n ");
}


__device__ void prefixScan(int * in_data, int * out_data, int n)
{

	// extern __shared__ float temp[];
	
	int thid = threadIdx.x;
	int offset = 1;

	for (int j = 0; j < numbersPerThread; j++) {
		out_data[j + thid * numbersPerThread] = in_data[j + thid * numbersPerThread];
	}



	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			out_data[bi] += out_data[ai];
		}
		offset *= 2;
	}
	if (thid == 0) { out_data[n - 1] = 0; }

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d)
		{
			int ai = offset*(2 * thid + 1) - 1;
			int bi = offset*(2 * thid + 2) - 1;
			float t = out_data[ai];
			out_data[ai] = out_data[bi];
			out_data[bi] += t;
		}
	}
	__syncthreads();


	//printf("Prefix scan %d ", threadIdx.x);	

}


__global__ void medianSplit(float *x, float *y, float * x_n, float * y_n, int * d_l, int * d_r)
{
    int i = threadIdx.x;
	int amountOfThreads = blockDim.x;

	int l = *d_l;
	int r = *d_r;
	//printf("R = %d \n" , r);
	//printf("L = %d \n", l);


	__shared__ float x_shared[numberOfPoints];
	__shared__ float y_shared[numberOfPoints];

	__shared__ float x_shared_tmp[numberOfPoints];
	__shared__ float y_shared_tmp[numberOfPoints];


	//__shared__ int position[numberOfPoints];

	__shared__ int pointsLessThanPivot;

	__shared__ int prefixBlock[numberOfPoints];
	__shared__ int prefixSumBlock[numberOfPoints];

	__shared__ int prefixBlockReverse[numberOfPoints];
	__shared__ int prefixSumBlockReverse[numberOfPoints];

	int prefixPerThread[numbersPerThread];

	for (int j = 0; j < numbersPerThread; j++) {
		x_shared[j + i * numbersPerThread] = x[j + i * numbersPerThread];
		y_shared[j + i * numbersPerThread] = y[j + i * numbersPerThread];
	//	position[j + i * numbersPerThread] = j + i * numbersPerThread;
	}
	
	int k_th = (l + r) >> 1;

	int pivot = (l + r) >> 1;

	//printf("pivot = %d \n", pivot);

	__syncthreads();
	while (k_th > 0) {

		if (!i) {
			//printf("pivot = [%.f, %.f]\n", x_shared[pivot], y_shared[pivot]);

			swap(x_shared[pivot], x_shared[r]);
			swap(y_shared[pivot], y_shared[r]);

			//swap(position[pivot], position[r]);

			//printf("pivot = [%.f, %.f]\n", x_shared[r], y_shared[r]);
		}
		__syncthreads();
		

		for (int j = 0; j < numbersPerThread; j++) {
			int i_index = j + i * numbersPerThread;
			//prefixPerThread[j] = x[i_index] < x[r] ? 1 : 0;
			prefixBlock[i_index] = x_shared[i_index] < x_shared[r] ? 1 : 0;
			prefixBlockReverse[i_index] = (prefixBlock[i_index] + 1) % 2;

			//printf("COMPARE x[i_index] < x[r] ? 1 : 0  = %f < %f = %d \n", x_shared[i_index], x_shared[r], prefixBlock[i_index]);
		}

		__syncthreads();
		
		if (!i) {
			printArray(prefixBlock, numberOfPoints);
		}

		prefixScan(prefixBlock, prefixSumBlock, numberOfPoints);
		prefixScan(prefixBlockReverse, prefixSumBlockReverse, numberOfPoints);


		if (!i) {
			pointsLessThanPivot = prefixSumBlock[numberOfPoints - 1];

			printArray(prefixSumBlock, numberOfPoints);
			printArray(prefixSumBlockReverse, numberOfPoints);
		}
		__syncthreads();

		for (int j = 0; j < numbersPerThread; j++) {
			int i_index = j + i * numbersPerThread;

			int new_index = prefixBlock[i_index] ? prefixSumBlock[i_index] : prefixSumBlockReverse[i_index] + pointsLessThanPivot;

			x_shared_tmp[new_index] = x_shared[i_index];
			y_shared_tmp[new_index] = y_shared[i_index];
		}

		__syncthreads();

		for (int j = 0; j < numbersPerThread; j++) {
			int i_index = j + i * numbersPerThread;

			x_shared[i_index] = x_shared_tmp[i_index];
			y_shared[i_index] = y_shared_tmp[i_index];
		}

		if (!i) {
			swap(x_shared[r], x_shared[pointsLessThanPivot]);
			swap(y_shared[r], y_shared[pointsLessThanPivot]);
		}

		k_th = 0;
	}

	__syncthreads();

	for (int j = 0; j < numbersPerThread; j++) {
		//x_n[position[j + i * numbersPerThread]] = x_shared[j + i * numbersPerThread];
		//y_n[position[j + i * numbersPerThread]] = y_shared[j + i * numbersPerThread];
		x_n[j + i * numbersPerThread] = x_shared[j + i * numbersPerThread];
		y_n[j + i * numbersPerThread] = y_shared[j + i * numbersPerThread];
	}

}

void printArray(float * x, float * y)
{
	for (int i = 0; i < numberOfPoints; i++) {
		printf("Point %d : [%.0f, %.0f]\n", i, x[i], y[i]);
	}
}

int main()
{
	
	KDTree tree(numberOfPoints, KDTree::BUILD_RECURSIVE_FAST, NULL, false);

	std::vector<glm::vec2> points = tree.p->getPointsPositions();

	float * h_x = new float[numberOfPoints];
	float * h_y = new float[numberOfPoints];
	float * h_x_n = new float[numberOfPoints];
	float * h_y_n = new float[numberOfPoints];


	for (int i = 0; i < numberOfPoints; i++) {
		h_x[i] = points[i].x;
		h_y[i] = points[i].y;
	}

	tree.setData();

	tree.printTree();


	// CUDA CODE

	int size = numberOfPoints;

	float *d_x = 0;
	float *d_y = 0;
	float *d_x_n = 0;
	float *d_y_n = 0;

	int * d_l = 0;
	int * d_r = 0;

	int * h_l = new int;
	int * h_r = new int;
	*h_r = numberOfPoints - 1;
	*h_l = 0;



	// Choose which GPU to run on, change this on a multi-GPU system.
	hipSetDevice(0);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	hipMalloc((void**)&d_x, size * sizeof(float));
	hipMalloc((void**)&d_y, size * sizeof(float));
	hipMalloc((void**)&d_x_n, size * sizeof(float));
	hipMalloc((void**)&d_y_n, size * sizeof(float));
	hipMalloc((void**)&d_l, sizeof(int));
	hipMalloc((void**)&d_r, sizeof(int));


	auto t0 = Time::now();
	
	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(d_x, h_x, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, h_y, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_l, h_l, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_r, h_r, sizeof(int), hipMemcpyHostToDevice);

	// Launch a kernel on the GPU with one thread for each element.
	medianSplit << <1, size/2 >> >(d_x, d_y, d_x_n, d_y_n, d_l, d_r);

	// Check for any errors launching the kernel
	hipGetLastError();

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
	hipMemcpy(h_x_n, d_x_n, size * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(h_y_n, d_y_n, size * sizeof(float), hipMemcpyDeviceToHost);

	auto t1 = Time::now();

	fsec fs = t1 - t0;
	ms d = std::chrono::duration_cast<ms>(fs);

	std::cout << "Time " << d.count() << "ms\n";

	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_x_n);
	hipFree(d_y_n);

	printArray(h_x, h_y);
	std::cout << "==============\n";
	printArray(h_x_n, h_y_n);
	/*

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	*/


	delete []h_x;
	delete []h_y;
	delete []h_x_n;
	delete []h_y_n;


	int tmp;
	std::cin >> tmp;
    
	return 0;
}
